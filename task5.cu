#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <ctime>
#include <string>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <iomanip>
#include "mpi.h"



__global__ void calculationMatrix(double* new_arry, const double* old_array, size_t size, size_t groupSize)
{
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    //printf("%d", size);
    if (i > 0 && i < groupSize - 1 && j > 0 && j < size - 1)
    {
        new_arry[i * size + j] = 0.25 * (old_array[i * size + j - 1] + old_array[(i - 1) * size + j] +
            old_array[(i + 1) * size + j] + old_array[i * size + j + 1]);
    }
}


__global__ void getDifferenceMatrix(const double* new_arry, const double* old_array, double* dif)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    dif[idx] = std::abs(old_array[idx] - new_arry[idx]);
}




int main(int argc, char** argv) {

    int SIZE;
    double err_max;
    int iter_max;

    SIZE = atoi(argv[2]);
    err_max = atof(argv[1]);
    iter_max = atoi(argv[3]);

    clock_t start;
    start = clock();

    int rank, sizeOfTheGroup;
    MPI_Init(&argc, &argv);
    //Функция возвращает номер процесса, вызвавшего эту функцию
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    // колво процессов в комуникаторе, запись в sizeOfTheGroup
    MPI_Comm_size(MPI_COMM_WORLD, &sizeOfTheGroup);


    hipSetDevice(rank);

    if (rank != 0)
        hipDeviceEnablePeerAccess(rank - 1, 0);
    if (rank != sizeOfTheGroup - 1)
        hipDeviceEnablePeerAccess(rank + 1, 0);

    size_t sizeOfAreaForOneProcess = SIZE / sizeOfTheGroup;
    size_t startYIdx = sizeOfAreaForOneProcess * rank;

 
    double* mas, * mas_old;
    hipHostMalloc(&mas, sizeof(double) * SIZE * SIZE);
    hipHostMalloc(&mas_old, sizeof(double) * SIZE * SIZE);

    std::memset(mas_old, 0, SIZE * SIZE * sizeof(double));


    mas[0] = 10;
    mas[SIZE - 1] = 20;
    mas[(SIZE) * (SIZE - 1)] = 20;
    mas[(SIZE) * (SIZE)-1] = 30;



    for (int i = 1; i < SIZE - 1; i++)
        mas[i] = mas[i - 1] + (mas[SIZE - 1] - mas[0]) / SIZE;

    for (int i = 1; i < SIZE - 1; i++) {
        mas[SIZE * (SIZE - 1) + i] = mas[SIZE * (SIZE - 1) + i - 1] + (mas[(SIZE) * (SIZE)-1] - mas[(SIZE) * (SIZE - 1)]) / SIZE;
        mas[(SIZE) * (i)] = mas[(i - 1) * (SIZE)] + (mas[(SIZE) * (SIZE - 1)] - mas[0]) / SIZE;
        mas[(SIZE) * (i)+(SIZE - 1)] = mas[(SIZE) * (i - 1) + (SIZE - 1)] + (mas[(SIZE) * (SIZE)-1] - mas[SIZE - 1]) / SIZE;
    }


    for (int i = 0; i < SIZE * SIZE; i++)
        mas_old[i] = mas[i];    



    int iter_count = 0; 
    double* error;
    hipHostMalloc(&error, sizeof(double));
    *error = 1.0;



    if (rank != 0 && rank != sizeOfTheGroup - 1)
    {
        sizeOfAreaForOneProcess += 2;
    }
    else
    {
        sizeOfAreaForOneProcess += 1;
    }

    size_t sizeOfAllocatedMemory = SIZE * sizeOfAreaForOneProcess;

    //выделяем память на gpu 
    double* d_mas, * d_mas_old, * d_dif;
    hipMalloc((void**)&d_mas_old, sizeof(double) * sizeOfAllocatedMemory);
    hipMalloc((void**)&d_mas, sizeof(double) * sizeOfAllocatedMemory);
    hipMalloc((void**)&d_dif, sizeof(double) * sizeOfAllocatedMemory);



    /// ///////////////////////
    unsigned int threads_x = (SIZE < 1024) ? SIZE : 1024;// кол-во потоков 
    unsigned int blocks_y = sizeOfAreaForOneProcess;// колво блоков 
    unsigned int blocks_x = SIZE / threads_x;

    dim3 blockDim1(threads_x, 1);//- размеры одного блока в потоках
    dim3 gridDim1(blocks_x, blocks_y);//размер сетки в блоках
    /// ////////////////////////////////


    size_t offset = (rank != 0) ? SIZE : 0;

    hipMemset(d_mas_old, 0, sizeof(double) * sizeOfAllocatedMemory);
    hipMemset(d_mas, 0, sizeof(double) * sizeOfAllocatedMemory);

    // CPU to GPU
    hipMemcpy(d_mas_old, mas_old + (startYIdx * SIZE) - offset, sizeof(double) * sizeOfAllocatedMemory, hipMemcpyHostToDevice); // (CPU) mas_old -> (GPU) d_mas_old
    hipMemcpy(d_mas, mas + (startYIdx * SIZE) - offset, sizeof(double) * sizeOfAllocatedMemory, hipMemcpyHostToDevice); // (CPU) mas -> (GPU) d_mas

  
    double* max_error = 0;
    hipMalloc((void**)&max_error, sizeof(double));




    size_t temp_storage_bytes = 0;
    double* temp_storage = NULL;
    //получаем размер временного буфера для редукции
    hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, d_dif, max_error, sizeOfAllocatedMemory);
    //выделяем память для буфера
    hipMalloc((void**)&temp_storage, temp_storage_bytes);

    hipStream_t stream;
    hipStreamCreate(&stream);


 

   
    while (iter_count < iter_max && (*error) > err_max) {
        iter_count += 1;
        // расчет матрицы
        calculationMatrix <<<gridDim1, blockDim1, 0, stream >>> (d_mas, d_mas_old, SIZE, sizeOfAreaForOneProcess);

     
        if (iter_count % 100 == 0) {

            getDifferenceMatrix <<<gridDim1, blockDim1, 0, stream >>> (d_mas, d_mas_old, d_dif);
            hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, d_dif, max_error, sizeOfAllocatedMemory); // нахождение максимума в разнице матрицы

            hipStreamSynchronize(stream);

            //Объединяет значения из всех процессов и распределяет результат обратно во все процессы.
            MPI_Allreduce((void*)max_error, (void*)max_error, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

            hipMemcpyAsync(error, max_error, sizeof(double), hipMemcpyDeviceToHost, stream); // запись ошибки в переменную на host
            // Находим максимальную ошибку среди всех и передаём её всем процессам

        }
        hipStreamSynchronize(stream);


        // Обмен "граничными" условиями каждой области
    // Обмен верхней границей
        if (rank != 0)
        {
            MPI_Sendrecv(d_mas + SIZE + 1, SIZE - 2, MPI_DOUBLE, rank - 1, 0, d_mas + 1, SIZE - 2, MPI_DOUBLE, rank - 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        }

        // Обмен нижней границей
        if (rank != sizeOfTheGroup - 1)
        {
            MPI_Sendrecv(d_mas + (sizeOfAreaForOneProcess - 2) * SIZE + 1,
                SIZE - 2, MPI_DOUBLE, rank + 1, 0,
                d_mas + (sizeOfAreaForOneProcess - 1) * SIZE + 1,
                SIZE - 2, MPI_DOUBLE, rank + 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        }

        double* c = d_mas_old;
        d_mas_old = d_mas;
        d_mas = c;
    }

    clock_t end = clock();
    if (rank == 0)
    {
        printf("%d %lf", (end - start) / CLOCKS_PER_SEC, &err);
    }

    //очитска памяти
    hipFree(d_mas_old);
    hipFree(d_mas);
    hipFree(temp_storage);
    hipFree(mas_old);
    hipFree(mas);
    hipFree(max_error);

    //Функция закрывает все MPI-процессы и ликвидирует все области связи
    MPI_Finalize();
        
    return 0;
}

