#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>


__global__
void countnewmatrix(double* mas_old, double* mas, size_t size)
{
	size_t i = blockIdx.x;
	size_t j = threadIdx.x;

	assert(i * size + j > size * size);
	if (!(blockIdx.x == 0 || threadIdx.x == 0))
		mas[i * size + j] = 0.25 * (mas_old[i * size + j - 1] + mas_old[(i - 1) * size + j] + mas_old[(i + 1) * size + j] + mas_old[i * size + j + 1]);
	
}
__global__
void finderr(double* mas_old, double* mas, double* outMatrix, size_t size)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	assert(i * size + j > size * size);
	if (!(blockIdx.x == 0 || threadIdx.x == 0))
		outMatrix[idx] = fabs(mas[idx] - mas_old[idx]);
	
}

int find_threads(int size){
	if (size%32==0){
		return size/1024, 1024;
		//t=1024;
		//b=size/1024;
	}
	else{
		return int(size/1024)+1, 1024;
		//t=1024;
		//b=int(size/1024)+1;
	}

	//return b,t;
}




int main(int argc, char** argv) {
	int SIZE;
	double err_max;
	int iter_max;

	SIZE = atoi(argv[2]);
	err_max = atof(argv[1]);
	iter_max = atoi(argv[3]);


	clock_t start;
	start = clock();


	double* mas = (double*)calloc(SIZE * SIZE, sizeof(double));
	double* mas_old = (double*)calloc(SIZE * SIZE, sizeof(double));

	mas[0] = 10;
	mas[SIZE - 1] = 20;
	mas[(SIZE) * (SIZE - 1)] = 20;
	mas[(SIZE) * (SIZE)-1] = 30;



	for (int i = 1; i < SIZE - 1; i++)
		mas[i] = mas[i - 1] + (mas[SIZE - 1] - mas[0]) / SIZE;

	for (int i = 1; i < SIZE - 1; i++) {
		mas[SIZE * (SIZE - 1) + i] = mas[SIZE * (SIZE - 1) + i - 1] + (mas[(SIZE) * (SIZE)-1] - mas[(SIZE) * (SIZE - 1)]) / SIZE;
		mas[(SIZE) * (i)] = mas[(i - 1) * (SIZE)] + (mas[(SIZE) * (SIZE - 1)] - mas[0]) / SIZE;
		mas[(SIZE) * (i)+(SIZE - 1)] = mas[(SIZE) * (i - 1) + (SIZE - 1)] + (mas[(SIZE) * (SIZE)-1] - mas[SIZE - 1]) / SIZE;
	}


	int iter = 0;
	double err = 1.0;


	for (int i = 0; i < SIZE * SIZE; i++) 
		mas_old[i] = mas[i];


	


	hipSetDevice(3);

	double* mas_old_dev, * mas_dev, * deviceError, * errorMatrix, * tempStorage = NULL; //Device-accessible allocation of temporary storage
	size_t tempStorageSize = 0;

	hipMalloc((void**)(&mas_old_dev), sizeof(double) * SIZE*SIZE);
	hipMalloc((void**)(&mas_dev), sizeof(double) * SIZE*SIZE);
	hipMalloc((void**)&deviceError, sizeof(double));
	hipMalloc((void**)&errorMatrix, sizeof(double) * SIZE*SIZE);

	hipMemcpy(mas_old_dev, mas_old, sizeof(double) * SIZE*SIZE, hipMemcpyHostToDevice);
	hipMemcpy(mas_dev, mas, sizeof(double) * SIZE*SIZE, hipMemcpyHostToDevice);

	// Determine temporary device storage requirements
	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, SIZE * SIZE);
	// Allocate temporary storage
	hipMalloc(&tempStorage, tempStorageSize);
	
	bool graphCreated=false;
	hipGraph_t graph;
	hipGraphExec_t instance;

		while ((err > err_max) && iter < iter_max) {
			iter += 1;

			int b,t = find_threads(SIZE);

			countnewmatrix <<<b, t>>> (mas_old_dev, mas_dev, SIZE);
			 if(!graphCreated){
    				hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
				if (iter % 100 == 0 && iter !=1)
				{
					finderr <<<b, t>>> (mas_old_dev, mas_dev, errorMatrix, SIZE);

					hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, SIZE*SIZE);
					hipMemcpy(&err, deviceError, sizeof(double), hipMemcpyDeviceToHost);

					double t = (double)(clock() - start) / CLOCKS_PER_SEC;
					printf(" time: %lf\n", t);
					printf("%d  %lf", iter, err);
					printf("\n");

				}
					double* m = mas_dev;
					mas_dev = mas_old_dev;
					mas_old_dev = m;
				hipStreamEndCapture(stream, &graph);
				hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
				graphCreated=true;

			}
			hipGraphLaunch(instance, stream);
  			hipStreamSynchronize(stream);
		}

	hipFree(mas_old_dev);
	hipFree(mas_dev);
	hipFree(errorMatrix);
	hipFree(tempStorage);
	//for (int i = 0; i < SIZE * SIZE; i++)
	free(mas_old);

	//for (int i = 0; i < SIZE*SIZE; i++)
	free(mas);

	double t = (double)(clock() - start) / CLOCKS_PER_SEC;
	printf(" time: %lf\n", t);
	return EXIT_SUCCESS;

}
