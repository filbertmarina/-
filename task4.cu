#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>


///////////////////////////////////////////////////
//подсчет новой матрицы на gpu
///////////////////////////////////////////////////
__global__
void countnewmatrix(double* mas_old, double* mas, size_t size)
{
	size_t i = blockIdx.x;
	size_t j = threadIdx.x;

	if (!(blockIdx.x == 0 || threadIdx.x == 0))
		mas[i * size + j] = 0.25 * (mas_old[i * size + j - 1] + mas_old[(i - 1) * size + j] + 
			mas_old[(i + 1) * size + j] + mas_old[i * size + j + 1]);
	
}
///////////////////////////////////////////////////
//подсчет матрицы ошибок на gpu
///////////////////////////////////////////////////
__global__
void finderr(double* mas_old, double* mas, double* outMatrix)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (!(blockIdx.x == 0 || threadIdx.x == 0))
		outMatrix[idx] = fabs(mas[idx] - mas_old[idx]);
	
}



///////////////////////////////////////////////////
// Фунция main
///////////////////////////////////////////////////
int main(int argc, char** argv) {

	//Установка параметров 
	int SIZE;
	double err_max;
	int iter_max;
	SIZE = atoi(argv[2]);
	err_max = atof(argv[1]);
	iter_max = atoi(argv[3]);

	//Запуск clock();
	clock_t start;
	start = clock();

	//////////////////////////////////
	//Заполнение массивов mas, mas_old, выделение памяти на cpu
	//////////////////////////////////
	double* mas = (double*)calloc(SIZE * SIZE, sizeof(double));
	double* mas_old = (double*)calloc(SIZE * SIZE, sizeof(double));

	mas[0] = 10;
	mas[SIZE - 1] = 20;
	mas[(SIZE) * (SIZE - 1)] = 20;
	mas[(SIZE) * (SIZE)-1] = 30;

	for (int i = 1; i < SIZE - 1; i++)
		mas[i] = mas[i - 1] + (mas[SIZE - 1] - mas[0]) / SIZE;

	for (int i = 1; i < SIZE - 1; i++) {
		mas[SIZE * (SIZE - 1) + i] = mas[SIZE * (SIZE - 1) + i - 1] + (mas[(SIZE) * (SIZE)-1] - mas[(SIZE) * (SIZE - 1)]) / SIZE;
		mas[(SIZE) * (i)] = mas[(i - 1) * (SIZE)] + (mas[(SIZE) * (SIZE - 1)] - mas[0]) / SIZE;
		mas[(SIZE) * (i)+(SIZE - 1)] = mas[(SIZE) * (i - 1) + (SIZE - 1)] + (mas[(SIZE) * (SIZE)-1] - mas[SIZE - 1]) / SIZE;
	}

	for (int i = 0; i < SIZE * SIZE; i++) 
		mas_old[i] = mas[i];

	//////////////////////////////////

	
	int iter = 0;
	double err = 1.0;

	//выбор устройства device
	hipSetDevice(3);

	double* mas_old_dev, * mas_dev, * deviceError, * errorMatrix, * tempStorage = NULL; //Device-accessible allocation of temporary storage
	size_t tempStorageSize = 0;

	/////////////////////////////////////////////////////////
	//allocating size bytes of linear memory on the device
	/////////////////////////////////////////////////////////
	hipMalloc((void**)(&mas_old_dev), sizeof(double) * SIZE*SIZE);
	hipMalloc((void**)(&mas_dev), sizeof(double) * SIZE*SIZE);
	hipMalloc((void**)&deviceError, sizeof(double));
	hipMalloc((void**)&errorMatrix, sizeof(double) * SIZE*SIZE);
	/////////////////////////////////////////////////////////

	///allocating a memory pointer on gpu fom cpu
	hipMemcpy(mas_old_dev, mas_old, sizeof(double) * SIZE*SIZE, hipMemcpyHostToDevice);
	hipMemcpy(mas_dev, mas, sizeof(double) * SIZE*SIZE, hipMemcpyHostToDevice);

	//determine temporary device storage requirements
	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, SIZE * SIZE);
	//allocating temporary storage
	hipMalloc(&tempStorage, tempStorageSize);


	///////////////////////////////////////////////////////////////////////
	//Основной цикл программы: проверка максимальных значений, нахождение вектора ошибок, нахождение максимальной ошибки,
	// перезапись mas в mas_old
	///////////////////////////////////////////////////////////////////////
		while ((err > err_max) && iter < iter_max) {
			iter += 1;
			countnewmatrix <<<SIZE - 1, SIZE - 1 >>> (mas_old_dev, mas_dev, SIZE);

			if (iter % 100 == 0 && iter !=1)
			{
				finderr <<<SIZE - 1, SIZE - 1>>> (mas_old_dev, mas_dev, errorMatrix);

				hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, SIZE*SIZE);
				//just from deviceError on gpu to err on cpu
				hipMemcpy(&err, deviceError, sizeof(double), hipMemcpyDeviceToHost);

				//вывод времени работы и ошибки 
				double t = (double)(clock() - start) / CLOCKS_PER_SEC;
				printf(" time: %lf\n", t);
				printf("%d  %lf", iter, err);
				printf("\n");

			}
			//перезапись массивов
				double* m = mas_dev;
				mas_dev = mas_old_dev;
				mas_old_dev = m;
			
		}
	////////////////////////////////////////////////

	//using hipFree to free gpu memory
	hipFree(mas_old_dev);
	hipFree(mas_dev);
	hipFree(errorMatrix);
	hipFree(tempStorage);

	//using free to free cpu memory
	free(mas_old);
	free(mas);

	//вывод общего времени работы программы, функции clock()
	double t = (double)(clock() - start) / CLOCKS_PER_SEC;
	printf(" time: %lf\n", t);
	return EXIT_SUCCESS;

}
